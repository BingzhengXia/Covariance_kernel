#include "../include/complex_covariance.cuh"
#include <hip/hip_complex.h>
#include <stdio.h>
#include <stdlib.h>
// 计算示例
int main() {
  // 设定样本数，每次计算一个样本批次的协方差矩阵，首先计算多个样本的均值，让后每个样本减去均值，然后计算协方差矩阵
  size_t num_samples = 10;

  hipFloatComplex *h_data = (hipFloatComplex *)malloc(num_samples * NUM_ELEMENTS *
                                                    sizeof(hipFloatComplex));
  hipFloatComplex *h_covariance_matrices = (hipFloatComplex *)malloc(
      num_samples * NUM_ELEMENTS * NUM_ELEMENTS * sizeof(hipFloatComplex));

  if (!h_data || !h_covariance_matrices) {
    fprintf(stderr, "主机内存分配失败\n");
    exit(EXIT_FAILURE);
  }

  // 初始化随机复数数据
  for (int i = 0; i < num_samples * NUM_ELEMENTS; i++) {
    h_data[i].x = (float)rand() / RAND_MAX * 2.0f - 1.0f;
    h_data[i].y = (float)rand() / RAND_MAX * 2.0f - 1.0f;
  }

  hipFloatComplex *d_data, *d_covariance_matrices;
  size_t data_size = num_samples * NUM_ELEMENTS * sizeof(hipFloatComplex);
  size_t covariance_size =
      num_samples * NUM_ELEMENTS * NUM_ELEMENTS * sizeof(hipFloatComplex);

  CHECK_CUDA_ERROR(hipMalloc((void **)&d_data, data_size));
  CHECK_CUDA_ERROR(
      hipMalloc((void **)&d_covariance_matrices, covariance_size));

  CHECK_CUDA_ERROR(
      hipMemcpy(d_data, h_data, data_size, hipMemcpyHostToDevice));

  // 调用设备指针版本的协方差计算函数
  computeComplexCovariance(d_data, num_samples, d_covariance_matrices);

  CHECK_CUDA_ERROR(hipMemcpy(h_covariance_matrices, d_covariance_matrices,
                              covariance_size, hipMemcpyDeviceToHost));

  // 打印部分结果用来验证
  printf("第一个样本协方差矩阵的前3x3部分:\n");
  for (int i = 0; i < 3; i++) {
    for (int j = 0; j < 3; j++) {
      printf("(%f, %f) ",
             h_covariance_matrices[0 * NUM_ELEMENTS * NUM_ELEMENTS +
                                   i * NUM_ELEMENTS + j]
                 .x,
             h_covariance_matrices[0 * NUM_ELEMENTS * NUM_ELEMENTS +
                                   i * NUM_ELEMENTS + j]
                 .y);
    }
    printf("\n");
  }

  if (num_samples > 1) {
    printf("\n第二个样本协方差矩阵的前3x3部分:\n");
    for (int i = 0; i < 3; i++) {
      for (int j = 0; j < 3; j++) {
        printf("(%f, %f) ",
               h_covariance_matrices[1 * NUM_ELEMENTS * NUM_ELEMENTS +
                                     i * NUM_ELEMENTS + j]
                   .x,
               h_covariance_matrices[1 * NUM_ELEMENTS * NUM_ELEMENTS +
                                     i * NUM_ELEMENTS + j]
                   .y);
      }
      printf("\n");
    }
  }
  
  free(h_data);
  free(h_covariance_matrices);
  CHECK_CUDA_ERROR(hipFree(d_data));
  CHECK_CUDA_ERROR(hipFree(d_covariance_matrices));

  return 0;
}